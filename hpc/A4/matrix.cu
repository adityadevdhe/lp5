#include "hip/hip_runtime.h"
%%writefile matrix_mult.cu
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;
#define BLOCK_SIZE 1
__global__ void gpuMM(float *A, float *B, float *C, int N) {
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
float sum = 0.f;
for (int n = 0; n < N; ++n)
sum += A[row * N + n] * B[n * N + col];
C[row * N + col] = sum;
}
int main(int argc, char *argv[]) {
int N;
// Get matrix size from user
cout << "Enter size of matrix (N): ";
cin >> N;
if (N % BLOCK_SIZE != 0) {
cerr << "Matrix size must be a multiple of BLOCK_SIZE." << endl;
return 1;
}
cout << "\nExecuting Matrix Multiplication" << endl;
cout << "Matrix size: " << N << "x" << N << endl;
// Allocate memory for matrices on the host
float *hA, *hB, *hC;
hA = new float[N * N];
hB = new float[N * N];
hC = new float[N * N];
// Read matrices from user
cout << "Enter elements of matrix A (" << N << "x" << N << "):" << endl;
for (int i = 0; i < N * N; ++i)
cin >> hA[i];
cout << "Enter elements of matrix B (" << N << "x" << N << "):" << endl;
for (int i = 0; i < N * N; ++i)
cin >> hB[i];
// Allocate memory for matrices on the device
int size = N * N * sizeof(float);
float *dA, *dB, *dC;
hipMalloc(&dA, size);
hipMalloc(&dB, size);
hipMalloc(&dC, size);
// Copy matrices from the host to the device
hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
dim3 threadBlock(BLOCK_SIZE, BLOCK_SIZE);
dim3 grid(N / BLOCK_SIZE, N / BLOCK_SIZE);
// Execute the matrix multiplication kernel
gpuMM<<<grid, threadBlock>>>(dA, dB, dC, N);
// Copy the result matrix from the device to the host
hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);
// Display the result matrix
cout << "\nResultant matrix:\n";
for (int row = 0; row < N; row++) {
for (int col = 0; col < N; col++) {
cout << hC[row * N + col] << " ";
}
cout << endl;
}
// Free device memory
hipFree(dA);
hipFree(dB);
hipFree(dC);
// Free host memory
delete[] hA;
delete[] hB;
delete[] hC;
cout << "Finished." << endl;
return 0;
}

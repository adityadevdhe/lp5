#include "hip/hip_runtime.h"
%%writefile add.cu
#include <iostream>
#include <cstdlib> // Include <cstdlib> for rand()
using namespace std;
__global__
void add(int* A, int* B, int* C, int size) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid < size) {
C[tid] = A[tid] + B[tid];
}
}
void print(int* vector, int size) {
for (int i = 0; i < size; i++) {
cout << vector[i] << " ";
}
cout << endl;
}
int main() {
int N;
cout << "Enter the size of the vectors: ";
cin >> N;
int* A, * B, * C;
int vectorSize = N;
size_t vectorBytes = vectorSize * sizeof(int);
// Allocate host memory
A = new int[vectorSize];
B = new int[vectorSize];
C = new int[vectorSize];
// Initialize host arrays
cout << "Enter elements of vector A:" << endl;
for (int i = 0; i < N; i++) {
cin >> A[i];
}
cout << "Enter elements of vector B:" << endl;
for (int i = 0; i < N; i++) {
cin >> B[i];
}
cout << "Vector A: ";
print(A, N);
cout << "Vector B: ";
print(B, N);
int* X, * Y, * Z;
// Allocate device memory
hipMalloc(&X, vectorBytes);
hipMalloc(&Y, vectorBytes);
hipMalloc(&Z, vectorBytes);
// Check for CUDA memory allocation errors
if (X == nullptr || Y == nullptr || Z == nullptr) {
cerr << "CUDA memory allocation failed" << endl;
return 1;
}
// Copy data from host to device
hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);
int threadsPerBlock = 256;
int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
// Launch kernel
add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);
// Check for kernel launch errors
hipError_t kernelLaunchError = hipGetLastError();
if (kernelLaunchError != hipSuccess) {
cerr << "CUDA kernel launch failed: " <<
hipGetErrorString(kernelLaunchError) << endl;
return 1;
}
// Copy result from device to host
hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);
// Check for CUDA memcpy errors
hipError_t memcpyError = hipGetLastError();
if (memcpyError != hipSuccess) {
cerr << "CUDA memcpy failed: " << hipGetErrorString(memcpyError) <<
endl;
return 1;
}
cout << "Addition: ";
print(C, N);
// Free device memory
hipFree(X);
hipFree(Y);
hipFree(Z);
// Free host memory
delete[] A;
delete[] B;
delete[] C;
return 0;
}
